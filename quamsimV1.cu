
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>




int count_lines(char *FILENAME)
{	
	int count=0;
	char c;
	FILE *FP = fopen(FILENAME, "r");
	while(c != EOF)
	{
		c = getc(FP);
		if(c == '\n')
			count++;
	}
	count++;
	fclose(FP);
	return count;
}


void setup_inputs(float **U, float **a, int *n, int *N, char *FILENAME)
{
	FILE *FP = fopen(FILENAME, "r");
	
	int lines_count = count_lines(FILENAME);
	char space;
	
	int U_len = 4;
	int a_len = lines_count - 5;
	*N = a_len;

	*U = (float *)malloc(U_len*sizeof(float));
	*a = (float *)malloc(a_len*sizeof(float));
	
	//read quantum gate
	fscanf(FP, "%f %f", *U, *U+1);
	fscanf(FP, "%f %f", *U+2, *U+3);

	//read blank
	fscanf(FP, "%c", space);

	// read a
	for(int i=0; i<a_len; i++)
		fscanf(FP,"%f", (*a+i));

	//read blank
	fscanf(FP, "%c", &space);

	// read n
	fscanf(FP, "%d", n);
	++*n;

      fclose(FP);

}





__global__ void qubit_gate(int *n, float *a, float *U, float *result, int *N)
{
	// global thread id
	int global_tid = (blockIdx.x * blockDim.x) + threadIdx.x;

	
	int src_index_1, src_index_2, two_pow_n, two_pow_n_minus_one;

	if(global_tid < *N/2)
	{	
		two_pow_n = 1 << *n;
		two_pow_n_minus_one = 1 << (*n-1);
		
		// source index mapped via global thread id
		src_index_1 = (global_tid % two_pow_n_minus_one) + (two_pow_n * (global_tid / two_pow_n_minus_one));
		src_index_2 = src_index_1 + two_pow_n_minus_one;

		//printf("\n threadid : %d, src_index_1 : %d, src_index_2 : %d", global_tid, src_index_1, src_index_2);
		// destination index mapped via global thread id
		int dest_index_1 = src_index_1;
		int dest_index_2 = src_index_2;

		// result computation based on thread id mapped indices
		result[dest_index_1] = (a[src_index_1] * U[0]) + (a[src_index_2] * U[1]);
		result[dest_index_2] = (a[src_index_1] * U[2]) + (a[src_index_2] * U[3]);
	}	

	
	
}

int main(int argc, char *argv[])
{

	if(argc != 2)
	{
		printf("\n arguments mismatch. exiting");
		exit(0);
	}

	char *IP_FILENAME = argv[1];

	// host copies 
	int n;
	int N;
	float *a, *U, *result;


	// setup input values
	// allocate spaces for host copies 

	setup_inputs( &U, &a, &n, &N, IP_FILENAME);
	
	int size_n = sizeof(int);
	int size_oprnds = N * sizeof(float);
	int size_U = 4 * sizeof(float);
	int size_N = sizeof(int);

	result = (float *)malloc(size_oprnds);	

	

	// device copies
	int *d_n, *d_N;
	float *d_a, *d_U, *d_result;
	

	//################### Device code #################################

	// timestamps
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
   

	// allocate spaces for device copies 
	hipMalloc((void **)&d_n,size_n);
	hipMalloc((void **)&d_a,size_oprnds);
	hipMalloc((void **)&d_result, size_oprnds);
	hipMalloc((void **)&d_U, size_U);
	hipMalloc((void **)&d_N, size_N);
	
	// copy inputs to device from host
	hipMemcpy(d_n, &n, size_n, hipMemcpyHostToDevice);
	hipMemcpy(d_a, a, size_oprnds, hipMemcpyHostToDevice);
	hipMemcpy(d_U, U, size_U, hipMemcpyHostToDevice);
	hipMemcpy(d_N, &N, size_N, hipMemcpyHostToDevice);
	
	
	// launch kernel
	dim3 threadsPerBlock(256,1);
	dim3 numBlocks(N/threadsPerBlock.x + 1,1);
	qubit_gate<<<numBlocks, threadsPerBlock>>>(d_n, d_a, d_U, d_result, d_N);
	
	hipEventRecord(stop);

	hipEventSynchronize(stop);

	// copy output to host from device
	hipMemcpy(result, d_result, size_oprnds, hipMemcpyDeviceToHost);
	
	//#################################################################
	float execute_time = 0;
	hipEventElapsedTime(&execute_time, start, stop);
	for(int i=0; i<N; i++)
           printf("%.3f\n", result[i]);
    //printf("\nExecution Time  : %f", execute_time);
	// cleanup
	hipFree(d_n);
	hipFree(d_a);
	hipFree(d_U);
	hipFree(d_result);
	free(result);
	free(a);
	free(U);
	
	return 0;
	

}
